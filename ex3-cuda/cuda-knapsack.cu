#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-knapsack.cu - Solve the 0/1 integer knapsack problem using CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * --------------------------------------------------------------------------
 *
 * Compile with:
 * nvcc cuda-knapsack.cu -o cuda-knapsack
 *
 * Run with:
 * ./cuda-knapsack < knapsack-100-1000.in
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>       /* for fmaxf() */
#include <assert.h>     /* for assert() */

/* Problem instance */
typedef struct {
  int C;          /* capacity             */
  int n;          /* number of items      */
  int *w;         /* array of n weights   */
  float *v;       /* array of n values    */
} knapsack_t;

/**
 * Given a set of n objects of weights w[0], ... w[n-1] and values
 * v[0], ... v[n-1], compute the maximum profit that can be obtained
 * by putting a subset of objects into a container of total capacity
 * C. Formally, the goal is to find a binary vector x[0], ... x[n-1]
 * such that:
 *
 * sum_{i=0}^{n-1} x[i] * v[i] is maximized
 *
 * subject to: sum_{i=0}^{n-1} x[i] * w[i] <= C
 *
 * This function uses the standard approach based on dynamic
 * programming; therefore, it requires space proportional to n*C
 */
float knapsack(int C, int n, int* w, float *v)
{
  const int NROWS = n;
  const int NCOLS = C+1;
  float *Pcur, *Pnext;
  float result;
  int i, j;

  /* [TODO] questi array andranno allocati nella memoria del device */    
  Pcur = (float*)malloc(NCOLS*sizeof(*Pcur)); assert(Pcur);
  Pnext = (float*)malloc(NCOLS*sizeof(*Pnext)); assert(Pnext);

  /* Inizializzazione: [TODO] volendo si puo' trasformare questo
     ciclo in un kernel CUDA, oppure si puo' far calcolare dalla CPU
     e successivamente trasferire Pcur nella memoria del device. */
  for (j=0; j<NCOLS; j++) {
    Pcur[j] = (j < w[0] ? 0.0 : v[0]);
  }
  /* Compute the DP matrix row-wise */
  for (i=1; i<NROWS; i++) {
    /* [TODO] Scrivere un kernel che esegua il ciclo seguente
       eseguendo "NCOLS" CUDA thread in parallelo */
    for (j=0; j<NCOLS; j++) {
      Pnext[j] = Pcur[j];
      if ( j>=w[i] ) {
        Pnext[j] = fmaxf(Pcur[j], Pcur[j - w[i]] + v[i]);
      }
    }
    /* Here, Pnext[j] is the maximum profit that can be obtained
       by putting a subset of items {0, 1, ... i} into a container
       of capacity j */
    float *tmp = Pcur;
    Pcur = Pnext;
    Pnext = tmp;
  }
  result = Pcur[NCOLS-1];
  free(Pcur); 
  free(Pnext);
  return result;
}

/* Read and allocate a problem instance from file |fin|; the file must
   conta, in order, C n w v. The problem instance can be deallocated
   with knapsack_free() */
void knapsack_load(FILE *fin, knapsack_t* k)
{
  int i;
  assert(k);
  fscanf(fin, "%d", &(k->C)); assert( k->C > 0 );
  fscanf(fin, "%d", &(k->n)); assert( k->n > 0 );
  k->w = (int*)malloc((k->n)*sizeof(int)); assert(k->w);
  k->v = (float*)malloc((k->n)*sizeof(float)); assert(k->v);
  for (i=0; i<(k->n); i++) {
    int nread = fscanf(fin, "%d %f", k->w + i, k->v + i);
    assert(2 == nread);
    assert(k->w[i] >= 0);
    assert(k->v[i] >= 0);
    /* fprintf(stderr, "%d %f\n", *(k->w + i), *(k->v + i)); */
  }
  fprintf(stderr, "Loaded instance with %d items, capacity %d\n", k->n, k->C);
}

/* Deallocate all memory used by a problem instance */
void knapsack_free(knapsack_t* k)
{
  assert(k);
  k->n = k->C = 0;
  free(k->w); k->w = NULL;
  free(k->v); k->v = NULL;
}

void knapsack_solve(const knapsack_t* k)
{
  assert(k);
  float result = knapsack(k->C, k->n, k->w, k->v);
  printf("Optimal profit: %f\n", result);
}

int main(int argc, char* argv[])
{
  knapsack_t k;

  if ( 1 != argc ) {
    fprintf(stderr, "Usage: %s < inputfile\n", argv[0]);
    return EXIT_FAILURE;
  }

  knapsack_load(stdin, &k);
  const double tstart = hpc_gettime();
  knapsack_solve(&k);
  const double elapsed = hpc_gettime() - tstart;
  /* Note that the execution time includes memory allocation and
     data movement to/from the GPU */
  fprintf(stderr, "Execution time: %f\n", elapsed);
  knapsack_free(&k);
  return EXIT_SUCCESS;
}
