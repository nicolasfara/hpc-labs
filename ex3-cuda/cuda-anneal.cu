#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-anneal.cu - Annealing with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * --------------------------------------------------------------------------
 *
 * CUDA implementaiton of the "ANNEAL" cellular automaton, with and
 * without shared memory.
 *
 * Compile with:
 * nvcc cuda-anneal.cu -o cuda-anneal
 *
 * Run with:
 * ./cuda-anneal [steps [n]]
 *
 * Example:
 * ./cuda-anneal 64
 * produces a file anneal-00064.pbm
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLKDIM_COPY 1024
#define BLKDIM      32


typedef unsigned char cell_t;

/* The following function makes indexing of the 2D domain
   easier. Instead of writing, e.g., grid[i*ext_n + j] you write
   IDX(grid, ext_n, i, j) to get a pointer to grid[i][j]. This
   function assumes that the size of the CA grid is (ext_n*ext_n),
   where the first and last rows/columns are ghost cells.

   Note the use of both the __device__ and __host__ qualifiers: this
   function can be called from host and device code. */
__device__ __host__ cell_t* IDX(cell_t *grid, int ext_n, int i, int j)
{
  return (grid + i*ext_n + j);
}

__device__ int d_min(int a, int b)
{
  return (a<b ? a : b);
}

/*
   |grid| points to a (ext_n * ext_n) block of bytes; this function
   copies the top and bottom ext_n elements to the opposite halo (see
   figure below).

   ext_n-2
   0 1              | ext_n-1
   | |              | |
   v v              v v
   +-+----------------+-+
   |Y|YYYYYYYYYYYYYYYY|Y| <- 0
   +-+----------------+-+
   |X|XXXXXXXXXXXXXXXX|X| <- 1
   | |                | |
   | |                | |
   | |                | |
   | |                | |
   |Y|YYYYYYYYYYYYYYYY|Y| <- ext_n - 2
   +-+----------------+-+
   |X|XXXXXXXXXXXXXXXX|X| <- ext_n - 1
   +-+----------------+-+

 */
/* [TODO] Transform this function into a kernel */
__global__ void copy_top_bottom(cell_t *grid, int ext_n)
{
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  if (j < ext_n) {
    *IDX(grid, ext_n, ext_n-1,j) = *IDX(grid, ext_n, 1, j); /* top to bottom halo */
    *IDX(grid, ext_n, 0, j)= *IDX(grid, ext_n, ext_n-2, j); /* bottom to top halo */
  }
}

/*
   |grid| points to a ext_n*ext_n block of bytes; this function copies
   the left and right ext_n elements to the opposite halo (see figure
   below).

   ext_n-2
   0 1              | ext_n-1
   | |              | |
   v v              v v
   +-+----------------+-+
   |Y|X              Y|X| <- 0
   +-+----------------+-+
   |Y|X              Y|X| <- 1
   |Y|X              Y|X|
   |Y|X              Y|X|
   |Y|X              Y|X|
   |Y|X              Y|X|
   |Y|X              Y|X| <- ext_n - 2
   +-+----------------+-+
   |Y|X              Y|X| <- ext_n - 1
   +-+----------------+-+

 */
/* [TODO] This function should be transformed into a kernel */
__global__ void copy_left_right(cell_t *grid, int ext_n)
{
  const int i = threadIdx.x + blockIdx.x + blockDim.x;
  if (i < ext_n) {
    *IDX(grid, ext_n, i, ext_n-1) = *IDX(grid, ext_n, i, 1); /* left column to right halo */
    *IDX(grid, ext_n, i, 0) = *IDX(grid, ext_n, i, ext_n-2); /* right column to left halo */
  }
}

/* Compute the |next| grid given the current configuration |cur|.
   Both grids have (ext_n*ext_n) elements.
   [TODO] This function should be transformed into a kernel. */
__global__ void step(cell_t *cur, cell_t *next, int ext_n)
{
  __shared__ cell_t buf[BLKDIM][BLKDIM];

  const int gj = threadIdx.x + blockIdx.x * (blockDim.x - 2);
  const int gi = threadIdx.y + blockIdx.y * (blockDim.y - 2);

  const int li = threadIdx.y;
  const int lj = threadIdx.x;

  if (gi < ext_n && gj < ext_n) {
    buf[li][lj] = *IDX(cur, ext_n, gi, gj);
    __syncthreads();
    if (li > 0 && li < blockDim.y - 1 && lj > 0 && lj < blockDim.x - 1) {
      const int nbors =
        buf[li - 1][lj - 1] + buf [li - 1][lj] + buf[li - 1][lj + 1] +
        buf[li][lj - 1] + buf[li][lj] + buf[li][lj + 1] +
        buf[li + 1][lj - 1] + buf[li + 1][lj] + buf[li + 1][lj + 1];
        *IDX(next, ext_n, gi, gj) = (nbors >= 6 || nbors == 4);
    }
  }

}

/* Initialize the current grid |cur| with alive cells with density
   |p|. */
void init( cell_t *cur, int ext_n, float p )
{
  int i, j;
  for (i=1; i<ext_n-1; i++) {
    for (j=1; j<ext_n-1; j++) {
      *IDX(cur, ext_n, i, j) = (((float)rand())/RAND_MAX < p);
    }
  }
}

/* Write |cur| to file |fname| in pbm (portable bitmap) format. */
void write_pbm( cell_t *cur, int ext_n, const char* fname )
{
  int i, j;
  FILE *f = fopen(fname, "w");
  if (!f) { 
    fprintf(stderr, "Cannot open %s for writing\n", fname);
    exit(EXIT_FAILURE);
  }
  fprintf(f, "P1\n");
  fprintf(f, "# produced by cuda-anneal.cu\n");
  fprintf(f, "%d %d\n", ext_n-2, ext_n-2);
  for (i=1; i<ext_n-1; i++) {
    for (j=1; j<ext_n-1; j++) {
      fprintf(f, "%d ", *IDX(cur, ext_n, i, j));
    }
    fprintf(f, "\n");
  }
  fclose(f);
}

int main( int argc, char* argv[] )
{
  char fname[128];
  cell_t *cur, *next;
  cell_t *d_cur, *d_next;
  int s, nsteps = 64, n = 256;

  if ( argc > 3 ) {
    fprintf(stderr, "Usage: %s [nsteps [n]]\n", argv[0]);
    return EXIT_FAILURE;
  }

  if ( argc > 1 ) {
    nsteps = atoi(argv[1]);
  }

  if ( argc > 2 ) {
    n = atoi(argv[2]);
    n = (n > 2048 ? 2048 : n); /* maximum image size is 2048 */
  }

  const int ext_n = n+2;
  const size_t ext_size = ext_n * ext_n * sizeof(cell_t);

  fprintf(stderr, "Anneal CA: steps=%d size=%d\n", nsteps, n);
  cur = (cell_t*)malloc(ext_size); assert(cur);
  next = (cell_t*)malloc(ext_size); assert(next);

  hipMalloc((void **)&d_cur, ext_size);
  hipMalloc((void **)&d_next, ext_size);

  init(cur, ext_n, 0.5);

  hipMemcpy(d_cur, cur, ext_size, hipMemcpyHostToDevice);

  dim3 block(BLKDIM, BLKDIM);
  dim3 grid((n + BLKDIM - 3) / (BLKDIM - 2), (n + BLKDIM - 3) / (BLKDIM - 2));
  const double tstart = hpc_gettime();
  for (s=0; s<nsteps; s++) {
    copy_top_bottom<<<(ext_n + BLKDIM_COPY - 1) / BLKDIM_COPY, BLKDIM_COPY>>>(d_cur, ext_n);
    copy_left_right<<<(ext_n + BLKDIM_COPY - 1) / BLKDIM_COPY, BLKDIM_COPY>>>(d_cur, ext_n);
    step<<<grid, block>>>(d_cur, d_next, ext_n);
    cell_t *tmp = d_cur;
    d_cur = d_next;
    d_next = tmp;
  }
  hipDeviceSynchronize();
  const double elapsed = hpc_gettime() - tstart;
  hipMemcpy(cur, d_cur, ext_size, hipMemcpyDeviceToHost);
  snprintf(fname, sizeof(fname), "anneal-%05d.pbm", s);
  write_pbm(cur, ext_n, fname);
  free(cur);
  free(next);
  hipFree(d_cur);
  hipFree(d_next);
  fprintf(stderr, "Execution time: %f\n", elapsed);

  return EXIT_SUCCESS;
}
