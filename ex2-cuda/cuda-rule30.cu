/* */
/****************************************************************************
 *
 * cuda-rule30.cu - Rule30 Cellular Automaton with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * --------------------------------------------------------------------------
 *
 * This program implements the "rule 30 CA" as described in
 * https://en.wikipedia.org/wiki/Rule_30 . This program uses the CPU
 * only; the task is to parallelize the computation so that the GPU is
 * used.
 *
 * Compile with:
 * nvcc cuda-rule30.cu -o cuda-rule30
 *
 * Run with:
 * /cuda-rule30 1024 1024
 *
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef unsigned char cell_t;

/**
 * Given the current state of the CA, compute the next state.  This
 * version requires that the |cur| and |next| arrays are extended with
 * ghost cells; therefore, |ext_n| is the length of |cur| and |next|
 * _including_ ghost cells.
 *
 *                             +----- ext_n-2
 *                             |   +- ext_n-1
 *   0   1                     V   V
 * +---+-------------------------+---+
 * |///|                         |///|
 * +---+-------------------------+---+
 *
 */
void step( cell_t *cur, cell_t *next, int ext_n )
{
    int i;
    for (i=1; i<ext_n-1; i++) {
        const cell_t left   = cur[i-1];
        const cell_t center = cur[i  ];
        const cell_t right  = cur[i+1];
        next[i] = 
            ( left && !center && !right) ||
            (!left && !center &&  right) ||
            (!left &&  center && !right) ||
            (!left &&  center &&  right);
    }
}

/**
 * Initialize the domain; all cells are 0, with the exception of a
 * single cell in the middle of the domain. |cur| points to an array
 * of length |ext_n|; the length includes two ghost cells.
 */
void init_domain( cell_t *cur, int ext_n )
{
    int i;
    for (i=0; i<ext_n; i++) {
        cur[i] = 0;
    }
    cur[ext_n/2] = 1;
}

/**
 * Dump the current state of the CA to PBM file |out|. |cur| points to
 * an array of length |ext_n| that includes two ghost cells.
 */
void dump_state( FILE *out, const cell_t *cur, int ext_n )
{
    int i;
    for (i=1; i < ext_n-1; i++) {
        fprintf(out, "%d ", cur[i]);
    }
    fprintf(out, "\n");
}

int main( int argc, char* argv[] )
{
    const char *outname = "rule30.pbm";
    FILE *out;
    int width = 1024, steps = 1024, s;    
    cell_t *cur, *next;
    
    if ( argc > 3 ) {
        fprintf(stderr, "Usage: %s [width [steps]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        width = atoi(argv[1]);
    }

    if ( argc > 2 ) {
        steps = atoi(argv[2]);
    }

    const int ext_width = width + 2;
    const size_t ext_size = ext_width * sizeof(*cur); /* includes ghost cells */
    
    /* Create the output file */
    out = fopen(outname, "w");
    if ( !out ) {
        fprintf(stderr, "FATAL: cannot create file \"%s\"\n", outname);
        return EXIT_FAILURE;
    }
    fprintf(out, "P1\n");
    fprintf(out, "# produced by %s %d %d\n", argv[0], width, steps);
    fprintf(out, "%d %d\n", width, steps);

    /* Allocate space for the cur[] and next[] arrays */
    cur = (cell_t*)malloc(ext_size);    
    next = (cell_t*)malloc(ext_size);

    /* Initialize the domain */
    init_domain(cur, ext_width);
    
    /* Evolve the CA */
    for (s=0; s<steps; s++) {

        /* Dump the current state */
        dump_state(out, cur, ext_width);

        /* Fill ghost cells */
        cur[ext_width-1] = cur[1];
        cur[0] = cur[ext_width-2];
        
        /* Compute next state */
        step(cur, next, ext_width);

        /* swap cur and next */
        cell_t *tmp = cur;
        cur = next;
        next = tmp;
    }
    
    free(cur);
    free(next);

    fclose(out);
    
    return EXIT_SUCCESS;
}
