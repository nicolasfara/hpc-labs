#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-matsum.cu - Dense matrix-matrix addition with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Matrix sum with CUDA.
 *
 * Compile with:
 * nvcc cuda-matsum.cu -o cuda-matsum -lm
 *
 * Run with:
 * ./cuda-matsum
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define BLKSIZE 32

__global__ void matsum( float *p, float *q, float *r, int n )
{
  /* [TODO] Modify the body of this function to
     - allocate memory on the device
     - copy p and q to the device
     - call an appropriate kernel
     - copy the result back from the device to the host
     - free memory on the device
   */
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n && j < n) {
    r[i*n + j] = p[i*n + j] + q[i*n + j];
  }
}

/* Initialize square matrix p of size nxn */
void fill( float *p, int n )
{
  int i, j, k=0;
  for (i=0; i<n; i++) {
    for (j=0; j<n; j++) {
      p[i*n+j] = k;
      k = (k+1) % 1000;
    }
  }
}

/* Check result */
int check( float *r, int n )
{
  int i, j, k = 0;
  for (i=0; i<n; i++) {
    for (j=0; j<n; j++) {
      if (fabsf(r[i*n+j] - 2.0*k) > 1e-5) {
        fprintf(stderr, "Check FAILED: r[%d][%d] = %f, expeted %f\n", i, j, r[i*n+j], 2.0*k);
        return 0;
      }
      k = (k+1) % 1000;
    }
  }
  printf("Check OK\n");
  return 1;
}

int main( int argc, char *argv[] ) 
{
  float *p, *q, *r;
  float *d_p, *d_q, *d_r;
  int n = 1024;
  const int max_n = 5000;

  if ( argc > 2 ) {
    fprintf(stderr, "Usage: %s [n]\n", argv[0]);
    return EXIT_FAILURE;
  }

  if ( argc > 1 ) {
    n = atoi(argv[1]);
  }

  if ( n > max_n ) {
    fprintf(stderr, "FATAL: the maximum allowed matrix size is %d\n", max_n);
    return EXIT_FAILURE;
  }

  const size_t size = n*n*sizeof(*p);

  /* Allocate space for p, q, r */
  p = (float*)malloc(size); assert(p);
  fill(p, n);
  q = (float*)malloc(size); assert(q);
  fill(q, n);
  r = (float*)malloc(size); assert(r);

  hipMalloc((void **)&d_p, size);
  hipMalloc((void **)&d_q, size);
  hipMalloc((void **)&d_r, size);

  hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
  hipMemcpy(d_q, q, size, hipMemcpyHostToDevice);

  const double tstart = hpc_gettime();
  dim3 blk((n + BLKSIZE - 1) / BLKSIZE, (n + BLKSIZE - 1) / BLKSIZE);
  dim3 thr(BLKSIZE, BLKSIZE);
  matsum<<<blk, thr>>>(d_p, d_q, d_r, n);
  const double elapsed = hpc_gettime() - tstart;

  hipMemcpy(r, d_r, size, hipMemcpyDeviceToHost);

  printf("Elapsed time (including data movement): %f\n", elapsed);

  /* Check result */
  check(r, n);

  /* Cleanup */
  free(p); free(q); free(r);
  hipFree(d_p);
  hipFree(d_q);
  hipFree(d_r);

  return EXIT_SUCCESS;
}
