#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-matsum.cu - Dense matrix-matrix addition with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Matrix sum with CUDA.
 *
 * Compile with:
 * nvcc cuda-matsum.cu -o cuda-matsum -lm
 *
 * Run with:
 * ./cuda-matsum
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>


void matsum( float *p, float *q, float *r, int n )
{
    /* [TODO] Modify the body of this function to
       - allocate memory on the device
       - copy p and q to the device
       - call an appropriate kernel
       - copy the result back from the device to the host
       - free memory on the device
    */
    int i, j;
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            r[i*n + j] = p[i*n + j] + q[i*n + j];
        }
    }
}

/* Initialize square matrix p of size nxn */
void fill( float *p, int n )
{
    int i, j, k=0;
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            p[i*n+j] = k;
            k = (k+1) % 1000;
        }
    }
}

/* Check result */
int check( float *r, int n )
{
    int i, j, k = 0;
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            if (fabsf(r[i*n+j] - 2.0*k) > 1e-5) {
                fprintf(stderr, "Check FAILED: r[%d][%d] = %f, expeted %f\n", i, j, r[i*n+j], 2.0*k);
                return 0;
            }
            k = (k+1) % 1000;
        }
    }
    printf("Check OK\n");
    return 1;
}

int main( int argc, char *argv[] ) 
{
    float *p, *q, *r;
    int n = 1024;
    const int max_n = 5000;
    
    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [n]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        n = atoi(argv[1]);
    }

    if ( n > max_n ) {
        fprintf(stderr, "FATAL: the maximum allowed matrix size is %d\n", max_n);
        return EXIT_FAILURE;
    }
    
    const size_t size = n*n*sizeof(*p);

    /* Allocate space for p, q, r */
    p = (float*)malloc(size); assert(p);
    fill(p, n);
    q = (float*)malloc(size); assert(q);
    fill(q, n);
    r = (float*)malloc(size); assert(r);

    const double tstart = hpc_gettime();
    matsum(p, q, r, n);
    const double elapsed = hpc_gettime() - tstart;

    printf("Elapsed time (including data movement): %f\n", elapsed);
    
    /* Check result */
    check(r, n);

    /* Cleanup */
    free(p); free(q); free(r);
    
    return EXIT_SUCCESS;
}
