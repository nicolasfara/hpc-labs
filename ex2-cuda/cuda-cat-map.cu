#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-cat-map.cu - Arnold's cat map with CUDA
 *
 * Written in 2016 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last updated in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 *
 * nvcc cuda-cat-map.cu -o cuda-cat-map
 *
 * Run with:
 *
 * ./cuda-cat-map k < input_file > output_file
 *
 * to compute the k-th iterate of the cat map.  Input and output files
 * are in PGM (Portable Graymap) format; see "man pgm" for details
 * (however, you do not need to know anything about the PGM formato;
 * functions are provided below to read and write a PGM file). The
 * file cat.pgm can be used as a test image.  Example:
 *
 * ./cuda-cat-map 100 < cat.pgm > cat.100.pgm
 *
 * See https://en.wikipedia.org/wiki/Arnold%27s_cat_map for an explanation
 * of the cat map.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#define BLKSIZE 32


typedef struct {
  int width;   /* Width of the image (in pixels) */
  int height;  /* Height of the image (in pixels) */
  int maxgrey; /* Don't care (used only by the PGM read/write routines) */
  unsigned char *bmap; /* buffer of width*height bytes; each byte represents a pixel */
} img_t;

/**
 * Read a PGM file from file |f|. This function is not very robust; it
 * may fail on perfectly legal PGM images, but works for the provided
 * cat.pgm file.
 */
void read_pgm( FILE *f, img_t* img )
{
  char buf[1024];
  const size_t BUFSIZE = sizeof(buf);
  char *s; 
  int nread;

  /* Get the file type (must be "P5") */
  s = fgets(buf, BUFSIZE, f);
  if (0 != strcmp(s, "P5\n")) {
    fprintf(stderr, "FATAL: wrong file type %s\n", buf);
    exit(EXIT_FAILURE);
  }
  /* Get any comment and ignore it; does not work if there are
     leading spaces in the comment line */
  do {
    s = fgets(buf, BUFSIZE, f);
  } while (s[0] == '#');
  sscanf(s, "%d %d", &(img->width), &(img->height));
  /* get maxgrey; must be less than or equal to 255 */
  s = fgets(buf, BUFSIZE, f);
  sscanf(s, "%d", &(img->maxgrey));
  if ( img->maxgrey > 255 ) {
    fprintf(stderr, "FATAL: maxgray > 255 (%d)\n", img->maxgrey);
    exit(EXIT_FAILURE);
  }
  /* Get the binary data */
  img->bmap = (unsigned char*)malloc((img->width)*(img->height));
  nread = fread(img->bmap, 1, (img->width)*(img->height), f);
  if ( (img->width)*(img->height) != nread ) {
    fprintf(stderr, "FATAL: error reading input: expecting %d bytes, got %d\n", (img->width)*(img->height), nread);
    exit(EXIT_FAILURE);
  }
}

/**
 * Write image |img| to file |f|
 */
void write_pgm( FILE *f, const img_t* img )
{
  fprintf(f, "P5\n");
  fprintf(f, "# produced by cuda-cat-map\n");
  fprintf(f, "%d %d\n", img->width, img->height);
  fprintf(f, "%d\n", img->maxgrey);
  fwrite(img->bmap, 1, (img->width)*(img->height), f);
}

/**
 * Free bitmap
 */
void free_pgm( img_t *img )
{
  img->width = img->height = img->maxgrey = -1;
  free(img->bmap);
  img->bmap = NULL;
}


/**
 * Compute the |k|-th iterate of the cat map for image |img|. The
 * width and height of the input image must be equal. This function
 * replaces the bitmap of |img| with the one resulting after ierating
 * |k| times the cat map. You need to allocate a temporary image, with
 * the same size of the original one, so that you read the pixel from
 * the "old" image and copy them to the "new" image (this is similar
 * to a stencil computation, as was discussed in class). After
 * applying the cat map to all pixel of the "old" image the role of
 * the two images is exchanged: the "new" image becomes the "old" one,
 * and vice-versa. At the end of the function, the temporary image
 * must be deallocated.
 */
__global__ void cat_map(unsigned char* cur, unsigned char *next, int N )
{

  /* [TODO] Modify the body of this function to allocate device memory,
     do the appropriate data transfer, and launch a kernel */

  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < N && y < N) {
    int xnext = (2*x+y) % N;
    int ynext = (x + y) % N;
    next[xnext + ynext*N] = cur[x+y*N];
  }
}

int main( int argc, char* argv[] )
{
  img_t bmap;
  unsigned char *cur, *next;
  int niter;

  if ( argc != 2 ) {
    fprintf(stderr, "Usage: %s niter < input_image > output_image\n", argv[0]);
    return EXIT_FAILURE;
  }
  niter = atoi(argv[1]);
  read_pgm(stdin, &bmap);
  if ( bmap.width != bmap.height ) {
    fprintf(stderr, "FATAL: width (%d) and height (%d) of the input image must be equal\n", bmap.width, bmap.height);
    return EXIT_FAILURE;
  }

  const int bytes = bmap.height * bmap.width;

  CudaSafeCall(hipMalloc((void **)&cur, bytes));
  CudaSafeCall(hipMalloc((void **)&next, bytes));
  CudaSafeCall(hipMemcpy(cur, bmap.bmap, bytes, hipMemcpyHostToDevice));

  const double tstart = hpc_gettime();
  dim3 blk((bmap.width + BLKSIZE - 1) / BLKSIZE, (bmap.width + BLKSIZE - 1) / BLKSIZE);
  dim3 thr(BLKSIZE, BLKSIZE);
  for (int i = 0; i < niter; i++) {
    cat_map<<<blk, thr>>>(cur, next, bmap.height);
    hipDeviceSynchronize();
    unsigned char *tmp;
    tmp = cur;
    cur = next;
    next = tmp;
  }
  CudaCheckError();
  const double elapsed = hpc_gettime() - tstart;

  CudaSafeCall(hipMemcpy(bmap.bmap, cur, bytes, hipMemcpyDeviceToHost));

  fprintf(stderr, "Execution time: %f\n", elapsed);
  write_pgm(stdout, &bmap);
  free_pgm(&bmap);
  hipFree(cur);
  hipFree(next);
  return EXIT_SUCCESS;
}
