#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-image-manipulation.c - Image manipulation with CUDA
 *
 * Written in 2018 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 *
 * nvcc cuda-image-manipulation.cu -o cuda-image-manipulation
 *
 * Run with:
 *
 * ./cuda-image-manipulation < input_file > output_file
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define BLKSIZE 32

typedef struct {
  int width;   /* Width of the image (in pixels) */
  int height;  /* Height of the image (in pixels) */
  int maxgrey; /* Don't care (used only by the PGM read/write routines) */
  unsigned char *bmap; /* buffer of width*height bytes; each byte represents a pixel */
} img_t;

/**
 * Read a PGM file from file |f|. This function is not very robust; it
 * may fail on perfectly legal PGM images.
 */
void read_pgm( FILE *f, img_t* img )
{
  char buf[1024];
  const size_t BUFSIZE = sizeof(buf);
  char *s; 
  int nread;

  /* Get the file type (must be "P5") */
  s = fgets(buf, BUFSIZE, f);
  if (0 != strcmp(s, "P5\n")) {
    fprintf(stderr, "FATAL: wrong file type %s\n", buf);
    exit(EXIT_FAILURE);
  }
  /* Get any comment and ignore it; does not work if there are
     leading spaces in the comment line */
  do {
    s = fgets(buf, BUFSIZE, f);
  } while (s[0] == '#');
  sscanf(s, "%d %d", &(img->width), &(img->height));
  /* get maxgrey; must be less than or equal to 255 */
  s = fgets(buf, BUFSIZE, f);
  sscanf(s, "%d", &(img->maxgrey));
  if ( img->maxgrey > 255 ) {
    fprintf(stderr, "FATAL: maxgray > 255 (%d)\n", img->maxgrey);
    exit(EXIT_FAILURE);
  }
  /* Get the binary data */
  img->bmap = (unsigned char*)malloc((img->width)*(img->height));
  nread = fread(img->bmap, 1, (img->width)*(img->height), f);
  if ( (img->width)*(img->height) != nread ) {
    fprintf(stderr, "FATAL: error reading input: expecting %d bytes, got %d\n", (img->width)*(img->height), nread);
    exit(EXIT_FAILURE);
  }
}

/**
 * Write image |img| to file |f|
 */
void write_pgm( FILE *f, const img_t* img )
{
  fprintf(f, "P5\n");
  fprintf(f, "# produced by cuda-image-manipulation\n");
  fprintf(f, "%d %d\n", img->width, img->height);
  fprintf(f, "%d\n", img->maxgrey);
  fwrite(img->bmap, 1, (img->width)*(img->height), f);
}

/**
 * Free bitmap and set fields to invalid values
 */
void free_pgm( img_t *img )
{
  img->width = img->height = img->maxgrey = -1;
  free(img->bmap);
  img->bmap = NULL;
}

/**
 * Rotate image |orig| of size nxn 90 degrees clockwise; new image
 * goes to |rotated|
 */
__global__ void rotate_clockwise( unsigned char *orig, unsigned char *rotated, int n )
{
  /* [TODO] Implement this kernel */
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;


  if (x < n && y < n) {
    rotated[(n - x - 1) * n + y] = orig[x * n + y];
    rotated[x * n + y] = orig[(n - x - 1) * n + y];
  }
}

/**
 * Flip image |orig| vertically; new image goes to |flipped|
 */
__global__ void vertical_flip( unsigned char *orig, unsigned char *flipped, int n )
{
  /* [TODO] Implement this kernel */
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;


  if (x < n && y < n) {
    flipped[(n - x - 1) * n + y] = orig[x * n + y];
    flipped[x * n + y] = orig[(n - x - 1) * n + y];
  }
}

/**
 * Flip image |orig| horizontally; new image goes to |flipped|
 */
__global__ void horizontal_flip( unsigned char *orig, unsigned char *flipped, int n )
{
  /* [TODO] Implement this kernel */
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;


  if (x < n && y < n) {
    flipped[(n - y - 1) * n + x] = orig[y * n + x];
    flipped[y * n + x] = orig[(n - y - 1) * n + x];
  }
}

int main( int argc, char* argv[] )
{
  img_t bmap;
  unsigned char *d_orig, *d_new, *tmp;
  double tstart, elapsed;
  int op;
  enum {
    OP_ROTATE = 1,
    OP_V_FLIP = 2,
    OP_H_FLIP = 4
  };

  if ( argc != 2 ) {
    fprintf(stderr, "Usage: %s op < input_file\nop = %d (rotate clockwise), %d (vertical flip), %d (horizontal flip)", argv[0], OP_ROTATE, OP_V_FLIP, OP_H_FLIP);
    return EXIT_FAILURE;
  }
  op = atoi(argv[1]);
  read_pgm(stdin, &bmap);
  if ( bmap.width != bmap.height ) {
    fprintf(stderr, "FATAL: width (%d) and height (%d) of the input image must be equal\n", bmap.width, bmap.height);
    return EXIT_FAILURE;
  }
  const int n = bmap.width;

  /* Allocate images on device */
  const size_t size = n*n;
  CudaSafeCall( hipMalloc((void **)&d_orig, size) );
  CudaSafeCall( hipMalloc((void **)&d_new, size) );

  /* Copy input to device */
  CudaSafeCall( hipMemcpy(d_orig, bmap.bmap, size, hipMemcpyHostToDevice) );
  const dim3 block(BLKSIZE, BLKSIZE);
  const dim3 grid((n + BLKSIZE - 1)/BLKSIZE, (n + BLKSIZE - 1)/BLKSIZE);

  tstart = hpc_gettime();
  if (op & OP_ROTATE) {
    fprintf(stderr, "Select 1\n");
    rotate_clockwise<<< grid, block >>>( d_orig, d_new, n);
    CudaCheckError();
    tmp = d_orig; d_orig = d_new; d_new = tmp;
  }
  if (op & OP_V_FLIP) {
    fprintf(stderr, "Select 2\n");
    vertical_flip<<< grid, block >>>( d_orig, d_new, n);
    CudaCheckError();
    tmp = d_orig; d_orig = d_new; d_new = tmp;
  }
  if (op & OP_H_FLIP) {
    fprintf(stderr, "Select 4\n");
    horizontal_flip<<< grid, block >>>( d_orig, d_new, n);
    CudaCheckError();
    tmp = d_orig; d_orig = d_new; d_new = tmp;
  }

  hipDeviceSynchronize();
  elapsed = hpc_gettime() - tstart;
  /* Copy output to host */
  hipMemcpy(bmap.bmap, d_orig, size, hipMemcpyDeviceToHost);
  fprintf(stderr, "Execution time: %f\n", elapsed);
  write_pgm(stdout, &bmap);
  free_pgm(&bmap);
  hipFree(d_orig);
  hipFree(d_new);
  return EXIT_SUCCESS;
}
