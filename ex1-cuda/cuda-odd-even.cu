#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-odd-even.cu - Odd-even sort with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 * nvcc cuda-odd-even.cu -o cuda-odd-even
 *
 * Run with:
 * ./cuda-odd-even [len]
 *
 * Example:
 * ./cuda-odd-even
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLKSIZE 512

/* if *a > *b, swap them. Otherwise do nothing */
void cmp_and_swap( int* a, int* b )
{
  if ( *a > *b ) {
    int tmp = *a;
    *a = *b;
    *b = tmp;
  }
}

/* Odd-even transposition sort */
void odd_even_step( int* v, int n, int phase )
{
  if ( phase % 2 == 0 ) {
    /* (even, odd) comparisons */
    for (int i=0; i<n-1; i += 2 ) {
      cmp_and_swap( &v[i], &v[i+1] );
    }
  } else {
    /* (odd, even) comparisons */
    for (int i=1; i<n-1; i += 2 ) {
      cmp_and_swap( &v[i], &v[i+1] );
    }
  }
}

/* Odd-even transposition sort */
__global__ void odd_even_step_bad( int* v, int n, int phase )
{
  const int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n - 1 && index % 2 == phase % 2) {
    if ( v[index] > v[index + 1] ) {
      int tmp = v[index];
      v[index] = v[index + 1];
      v[index + 1] = tmp;
    }
  }
}

/* Odd-even transposition sort */
__global__ void odd_even_step_good( int* v, int n, int phase )
{
  const int index = (threadIdx.x + blockIdx.x * blockDim.x);
  const int i = 2 * index + (phase % 2);
  if (i < n - 1) {
    if ( v[i] > v[i + 1] ) {
      int tmp = v[i];
      v[i] = v[i + 1];
      v[i + 1] = tmp;
    }
  }
}

/**
 * Return a random integer in the range [a..b]
 */
int randab(int a, int b)
{
  return a + (rand() % (b-a+1));
}

/**
 * Fill vector x with a random permutation of the integers 0..n-1
 */
void fill( int *x, int n )
{
  int i, j, tmp;
  for (i=0; i<n; i++) {
    x[i] = i;
  }
  for(i=0; i<n-1; i++) {
    j = randab(i, n-1);
    tmp = x[i];
    x[i] = x[j];
    x[j] = tmp;
  }
}

/**
 * Check correctness of the result
 */
int check( int *x, int n )
{
  int i;
  for (i=0; i<n; i++) {
    if (x[i] != i) {
      fprintf(stderr, "Check FAILED: x[%d]=%d, expected %d\n", i, x[i], i);
      return 0;
    }
  }
  printf("Check OK\n");
  return 1;
}

int main( int argc, char *argv[] ) 
{
  int *x, *d_x;
  int phase, n = 128*1024;
  const int max_len = 512*1024*1024;
  double tstart, elapsed;

  if ( argc > 2 ) {
    fprintf(stderr, "Usage: %s [len]\n", argv[0]);
    return EXIT_FAILURE;
  }

  if ( argc > 1 ) {
    n = atoi(argv[1]);
  }

  if ( n > max_len ) {
    fprintf(stderr, "FATAL: the maximum length is %d\n", max_len);
    return EXIT_FAILURE;
  }

  const size_t size = n * sizeof(*x);

  /* Allocate space for x on host */
  x = (int*)malloc(size); assert(x);
  fill(x, n);

  hipMalloc((void **)&d_x, size);
  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

  tstart = hpc_gettime();
  for (phase = 0; phase < n; phase++) {
    //odd_even_step_bad<<<(n + BLKSIZE - 1) / BLKSIZE, BLKSIZE>>>(d_x, n, phase);
    odd_even_step_good<<<(n/2 + BLKSIZE - 1) / BLKSIZE, BLKSIZE>>>(d_x, n, phase);
  }
  hipDeviceSynchronize();

  elapsed = hpc_gettime() - tstart;
  printf("Sorted %d elements in %f seconds\n", n, elapsed);

  hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

  /* Check result */
  check(x, n);

  /* Cleanup */
  hipFree(d_x);
  free(x);

  return EXIT_SUCCESS;
}
