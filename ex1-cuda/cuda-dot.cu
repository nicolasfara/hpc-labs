#include "hip/hip_runtime.h"
/* */
/****************************************************************************
 *
 * cuda-dot.cu - Dot product with CUDA
 *
 * Written in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last modified in 2018 by Moreno Marzolla
 *
 * To the extent possible under law, the author(s) have dedicated all 
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * You should have received a copy of the CC0 Public Domain Dedication
 * along with this software. If not, see 
 * <http://creativecommons.org/publicdomain/zero/1.0/>. 
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 * nvcc cuda-dot.cu -o cuda-dot -lm
 *
 * Run with:
 * ./cuda-dot [len]
 *
 * Example:
 * ./cuda-dot
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>


double dot( double *x, double *y, int n )
{
    /* [TODO] modify this function so that (part of) the dot product
       computation is executed on the GPU. */
    double result = 0.0;
    for (int i = 0; i < n; i++) {
        result += x[i] * y[i];
    }
    return result;
}

void vec_init( double *x, double *y, int n )
{
    int i;
    const double tx[] = {1.0/64.0, 1.0/128.0, 1.0/256.0};
    const double ty[] = {1.0, 2.0, 4.0};
    const size_t arrlen = sizeof(tx)/sizeof(tx[0]);

    for (i=0; i<n; i++) {
        x[i] = tx[i % arrlen];
        y[i] = ty[i % arrlen];
    }
}

int main( int argc, char* argv[] ) 
{
    double *x, *y, result;
    int n = 1024*1024;
    const int max_len = 128 * n;

    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [len]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        n = atoi(argv[1]);
    }

    if ( n > max_len ) {
        fprintf(stderr, "FATAL: the maximum length is %d\n", max_len);
        return EXIT_FAILURE;
    }

    const size_t size = n*sizeof(*x);

    /* Allocate space for host copies of x, y */
    x = (double*)malloc(size); assert(x);
    y = (double*)malloc(size); assert(y);
    vec_init(x, y, n);

    printf("Computing the dot product of %d elements... ", n);
    result = dot(x, y, n);
    printf("result=%f\n", result);

    const double expected = ((double)n)/64;

    /* Check result */
    if ( fabs(result - expected) < 1e-5 ) {
        printf("Check OK\n");
    } else {
        printf("Check FAILED: got %f, expected %f\n", result, expected);
    }
    
    /* Cleanup */
    free(x); free(y);
    
    return EXIT_SUCCESS;
}
